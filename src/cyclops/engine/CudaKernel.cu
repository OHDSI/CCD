#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <hip/hip_vector_types.h>

#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/sequence.h>
#include <thrust/iterator/permutation_iterator.h>
//#include <thrust/transform_reduce.h>
//#include <thrust/for_each.h>

#include "CudaKernel.h"

using namespace hipcub;


template <typename RealType, FormatTypeCuda formatType>
__global__ void kernelComputeNumeratorForGradient(int offX,
                                                  int offK,
                                                  const int taskCount,
                                                  const RealType* d_X,
                                                  const int* d_K,
						  RealType* d_KWeight,
                                                  RealType* d_ExpXBeta,
                                                  RealType* d_Numerator,
                                                  RealType* d_Numerator2)
{
        int task = blockIdx.x * blockDim.x + threadIdx.x;

        int k;
        if (formatType == INDICATOR || formatType == SPARSE) {
            k = d_K[offK + task];
        } else { // DENSE, INTERCEPT
            k = task;
        }

        if (task < taskCount) {
            if (formatType == SPARSE || formatType == DENSE) {
                d_Numerator[k] = d_X[offX + task] * d_ExpXBeta[k] * d_KWeight[k];
                d_Numerator2[k] = d_X[offX + task] * d_Numerator[k];
            } else { // INDICATOR, INTERCEPT
                d_Numerator[k] = d_ExpXBeta[k] * d_KWeight[k];
            }
        }
}

template <typename RealType, PriorTypeCuda priorType>
__global__ void kernelProcessDelta(double2* d_GH,
		RealType* d_XjY,
		RealType* d_Delta,
		RealType* d_Beta,
		RealType* d_Bound,
		RealType* d_PriorParams,
		int index)
{
	// get gradient, hessian, and old beta
	double2 GH = *d_GH;
	RealType g = GH.x - d_XjY[index];
	RealType h = GH.y;
	RealType beta = d_Beta[index];
	
	// process delta according to prior type
	RealType delta;
	if (priorType == NOPRIOR) {
		delta = -g/h;
	}
	if (priorType == LAPLACE) {
		RealType lambda = d_PriorParams[index];
		RealType neg_update = - (g - lambda) / h;
		RealType pos_update = - (g + lambda) / h;
		if (beta == 0) {
			if (neg_update < 0) {
				delta = neg_update;
			} else if (pos_update > 0) {
				delta = pos_update;
			} else {
				delta = 0;
			}
		} else {
			if (beta < 0) {
				delta = neg_update;
				if (beta+delta > 0) delta = -beta;
			} else {
				delta = pos_update;
				if (beta+delta < 0) delta = -beta;
			}
		}
	}
	if (priorType == NORMAL) {
		RealType variance = d_PriorParams[index];
		delta = - (g + (beta / variance)) / (h + (1.0 / variance));
	}
	
	// update delta and beta
	RealType bound = d_Bound[index];
	if (delta < -bound) {
		delta = -bound;
	} else if (delta > bound) {
		delta = bound;
	}
	d_Delta[index] = delta;
	d_Beta[index] = delta + beta;
	
	// update bound
	auto intermediate = max(2*abs(delta), bound/2);
	intermediate = max(intermediate, 0.001);
	d_Bound[index] = intermediate;
}

template <typename RealType, FormatTypeCuda formatType>
__global__ void kernelUpdateXBeta(int offX,
		int offK,
		const int taskCount,
		RealType delta,
		const RealType* d_X,
		const int* d_K,
		RealType* d_KWeight,
		RealType* d_XBeta,
		RealType* d_ExpXBeta,
		RealType* d_Denominator,
		RealType* d_Numerator,
		RealType* d_Numerator2)
{
	// update xb, exb, and denom if needed
	// zero numer and numer2

	int task = blockIdx.x * blockDim.x + threadIdx.x;

	int k;
	if (formatType == INDICATOR || formatType == SPARSE) {
		k = d_K[offK + task];
	} else { // DENSE, INTERCEPT
		k = task;
	}

	if (delta != 0.0) { // update xb and exb, zero numer

		RealType inc;
		if (formatType == SPARSE || formatType == DENSE) {
			inc = delta * d_X[offX + task];
		} else { // INDICATOR, INTERCEPT
			inc = delta;
		}

		if (task < taskCount) {
			RealType xb = d_XBeta[k] + inc;
			d_XBeta[k] = xb;
			d_ExpXBeta[k] = exp(xb);
			d_Denominator[k] = exp(xb) * d_KWeight[k];
			d_Numerator[k] = 0;
			if (formatType != INDICATOR) {
				d_Numerator2[k] = 0;
			}
		}

	} else { // only zero numer

		if (task < taskCount) {
			d_Numerator[k] = 0;
			if (formatType != INDICATOR) {
				d_Numerator2[k] = 0;
			}
		}
	}
}


template <typename RealType, FormatTypeCuda formatType, PriorTypeCuda priorType>
__global__ void kernelUpdateXBetaAndDelta(int offX,
				  int offK,
				  const int taskCount,
				  int index,
				  const RealType* d_X,
				  const int* d_K,
				  double2* d_GH,
				  RealType* d_XjY,
				  RealType* d_Bound,
				  RealType* d_KWeight,
				  RealType* d_Beta,
				  RealType* d_XBeta,
				  RealType* d_ExpXBeta,
				  RealType* d_Denominator,
				  RealType* d_Numerator,
				  RealType* d_Numerator2,
				  RealType* d_PriorParams)
{
	// get gradient, hessian, and old beta
	double2 GH = *d_GH;
	RealType g = GH.x - d_XjY[index];
	RealType h = GH.y;
	RealType beta = d_Beta[index];

	// process delta
	RealType delta;
	if (priorType == NOPRIOR) {
		delta = -g/h;
	}
	if (priorType == LAPLACE) {
		RealType lambda = d_PriorParams[index];
		RealType neg_update = - (g - lambda) / h;
		RealType pos_update = - (g + lambda) / h;
		if (beta == 0) {
			if (neg_update < 0) {
				delta = neg_update;
			} else if (pos_update > 0) {
				delta = pos_update;
			} else {
				delta = 0;
			}
		} else {
			if (beta < 0) {
				delta = neg_update;
				if (beta+delta > 0) delta = -beta;
			} else {
				delta = pos_update;
				if (beta+delta < 0) delta = -beta;
			}
		}
	}
	if (priorType == NORMAL) {
		RealType variance = d_PriorParams[index];
		delta = - (g + (beta / variance)) / (h + (1.0 / variance));
	}

	// update beta
	RealType bound = d_Bound[index];
	if (delta < -bound) {
		delta = -bound;
	} else if (delta > bound) {
		delta = bound;
	}
	d_Beta[index] = delta + beta; // TODO: need grid-wide synchronization (execution) barrier

	// update bound
	auto intermediate = max(2*abs(delta), bound/2);
	intermediate = max(intermediate, 0.001);
	d_Bound[index] = intermediate;


	// update xb, exb, and denom if needed
	// zero numer and numer2

	int task = blockIdx.x * blockDim.x + threadIdx.x;
                
	int k;
	if (formatType == INDICATOR || formatType == SPARSE) {
		k = d_K[offK + task];
	} else { // DENSE, INTERCEPT
		k = task;
	}

	if (delta != 0.0) { // update xb and exb, zero numer
		
		RealType inc;
		if (formatType == SPARSE || formatType == DENSE) {
			inc = delta * d_X[offX + task];
		} else { // INDICATOR, INTERCEPT
			inc = delta;
		}
		
		if (task < taskCount) {
			RealType xb = d_XBeta[k] + inc;
			d_XBeta[k] = xb;
			d_ExpXBeta[k] = exp(xb);
			d_Denominator[k] = exp(xb) * d_KWeight[k];
			d_Numerator[k] = 0;
			if (formatType != INDICATOR) {
				d_Numerator2[k] = 0;
			}
		}

	} else { // only zero numer

		if (task < taskCount) {
			d_Numerator[k] = 0;
			if (formatType != INDICATOR) {
				d_Numerator2[k] = 0;
			}
		}
	}
}


struct TuplePlus
{
	template<typename L, typename R>
	__host__ __device__
	thrust::tuple<L, L> operator()(thrust::tuple<L, L> lhs, thrust::tuple<R, R> rhs)
	{
		return thrust::make_tuple(thrust::get<0>(lhs) + thrust::get<0>(rhs), thrust::get<1>(lhs) + thrust::get<1>(rhs));
	}
};

struct TuplePlus3
{
	template<typename L, typename R>
	__host__ __device__
	thrust::tuple<L, L, L> operator()(thrust::tuple<L, L, L> lhs, thrust::tuple<R, R, R> rhs)
	{
		return thrust::make_tuple(thrust::get<0>(lhs) + thrust::get<0>(rhs),
							thrust::get<1>(lhs) + thrust::get<1>(rhs),
							thrust::get<2>(lhs) + thrust::get<2>(rhs));
	}
};

struct Double2Plus
{
	__host__ __device__
	double2 operator()(double2& a, double2& b)
	{
		double2 out;
		out.x = a.x + b.x;
		out.y = a.y + b.y;
		return out;
	}
};


template <typename RealType>
CudaKernel<RealType>::CudaKernel()
{
	std::cout << "ctor CudaKernel \n";
}

template <typename RealType>
CudaKernel<RealType>::~CudaKernel()
{
	hipFree(d_temp_storage0); // accDenom
	hipFree(d_temp_storage_gh); // cGAH
//	hipFree(d_init);
	std::cout << "dtor CudaKernel \n";
}


template <typename RealType>
void CudaKernel<RealType>::allocTempStorage(thrust::device_vector<RealType>& d_Denominator,
					    thrust::device_vector<RealType>& d_Numerator,
					    thrust::device_vector<RealType>& d_Numerator2,
					    thrust::device_vector<RealType>& d_AccDenom,
					    thrust::device_vector<RealType>& d_AccNumer,
					    thrust::device_vector<RealType>& d_AccNumer2,
					    thrust::device_vector<RealType>& d_NWeight,
					    double2* d_GH,
					    double2* d_BlockGH,
					    size_t& N,
					    thrust::device_vector<int>& indicesN)
{
//	thrust::sequence(indicesN.begin(), indicesN.end());

	// for scan in accDenom
	DeviceScan::InclusiveSum(d_temp_storage0, temp_storage_bytes0, &d_Denominator[0], &d_AccDenom[0], N);
	hipMalloc(&d_temp_storage0, temp_storage_bytes0);
/*
	// for fused scan reduction
	auto begin0 = thrust::make_zip_iterator(thrust::make_tuple(d_Numerator.begin(), d_Numerator2.begin()));
	auto begin1 = thrust::make_zip_iterator(thrust::make_tuple(d_AccDenom.begin(), d_NWeight.begin()));
	DeviceFuse::ScanReduce(d_temp_storage_gh, temp_storage_bytes_gh, begin0, begin1, d_BlockGH, d_GH,
			TuplePlus(), Double2Plus(), compGradHessInd, N);
*/

	auto begin2 = thrust::make_zip_iterator(thrust::make_tuple(d_Numerator.begin(),
				d_Numerator2.begin(),
				d_Denominator.begin()));
	
	// triple scan without storing accDenom
	DeviceFuse::ScanReduce(d_temp_storage_gh, temp_storage_bytes_gh, 
			begin2, thrust::raw_pointer_cast(&d_NWeight[0]), 
			d_BlockGH, d_GH,
                        TuplePlus3(), Double2Plus(), compGradHessInd1, N);
/*
	// triple scan with storing accDenom
	DeviceFuse::ScanReduce1(d_temp_storage_gh, temp_storage_bytes_gh,
			begin2, thrust::raw_pointer_cast(&d_NWeight[0]),
			d_BlockGH, d_GH, thrust::raw_pointer_cast(&d_AccDenom[0]),
			TuplePlus3(), Double2Plus(), compGradHessInd1, scanOutput, N);
*/
	hipMalloc(&d_temp_storage_gh, temp_storage_bytes_gh);
}


template <typename RealType>
void CudaKernel<RealType>::computeNumeratorForGradient(const thrust::device_vector<RealType>& d_X,
						const thrust::device_vector<int>& d_K,
						unsigned int offX,
						unsigned int offK,
						const unsigned int taskCount,
						thrust::device_vector<RealType>& d_KWeight,
						thrust::device_vector<RealType>& d_ExpXBeta,
						thrust::device_vector<RealType>& d_Numerator,
						thrust::device_vector<RealType>& d_Numerator2,
						FormatType& formatType,
						int gridSize, int blockSize)
{
        switch (formatType) {
                case DENSE :
                        kernelComputeNumeratorForGradient<RealType, DENSE><<<gridSize, blockSize>>>(offX,
                                                                                               offK,
                                                                                               taskCount,
                                                                                               thrust::raw_pointer_cast(&d_X[0]),
                                                                                               thrust::raw_pointer_cast(&d_K[0]),
											       thrust::raw_pointer_cast(&d_KWeight[0]),
                                                                                               thrust::raw_pointer_cast(&d_ExpXBeta[0]),
                                                                                               thrust::raw_pointer_cast(&d_Numerator[0]),
                                                                                               thrust::raw_pointer_cast(&d_Numerator2[0]));
                break;	
                case SPARSE :
                        kernelComputeNumeratorForGradient<RealType, SPARSE><<<gridSize, blockSize>>>(offX,
                                                                                               offK,
                                                                                               taskCount,
                                                                                               thrust::raw_pointer_cast(&d_X[0]),
                                                                                               thrust::raw_pointer_cast(&d_K[0]),
											       thrust::raw_pointer_cast(&d_KWeight[0]),
                                                                                               thrust::raw_pointer_cast(&d_ExpXBeta[0]),
                                                                                               thrust::raw_pointer_cast(&d_Numerator[0]),
                                                                                               thrust::raw_pointer_cast(&d_Numerator2[0]));
                break;
                case INDICATOR :
                        kernelComputeNumeratorForGradient<RealType, INDICATOR><<<gridSize, blockSize>>>(offX,
                                                                                               offK,
                                                                                               taskCount,
                                                                                               thrust::raw_pointer_cast(&d_X[0]),
                                                                                               thrust::raw_pointer_cast(&d_K[0]),
											       thrust::raw_pointer_cast(&d_KWeight[0]),
                                                                                               thrust::raw_pointer_cast(&d_ExpXBeta[0]),
                                                                                               thrust::raw_pointer_cast(&d_Numerator[0]),
                                                                                               thrust::raw_pointer_cast(&d_Numerator2[0]));
                break;
                case INTERCEPT :
                        kernelComputeNumeratorForGradient<RealType, INTERCEPT><<<gridSize, blockSize>>>(offX,
                                                                                               offK,
                                                                                               taskCount,
                                                                                               thrust::raw_pointer_cast(&d_X[0]),
                                                                                               thrust::raw_pointer_cast(&d_K[0]),
											       thrust::raw_pointer_cast(&d_KWeight[0]),
                                                                                               thrust::raw_pointer_cast(&d_ExpXBeta[0]),
                                                                                               thrust::raw_pointer_cast(&d_Numerator[0]),
                                                                                               thrust::raw_pointer_cast(&d_Numerator2[0]));
                break;
	}
	
	hipDeviceSynchronize(); // MAS Wait until kernel completes; may be important for timing
}


template <typename RealType>
void CudaKernel<RealType>::computeGradientAndHessian(thrust::device_vector<RealType>& d_Numerator,
						     thrust::device_vector<RealType>& d_Numerator2,
						     thrust::device_vector<RealType>& d_AccNumer,
						     thrust::device_vector<RealType>& d_AccNumer2,
						     thrust::device_vector<RealType>& d_AccDenom,
						     thrust::device_vector<RealType>& d_NWeight,
						     double2* d_GH,
						     double2* d_BlockGH,
						     FormatType& formatType,
						     size_t& N
//						     ,const std::vector<int>& K,
//                                                     unsigned int offK,
//                                                     thrust::device_vector<int>& indicesN
						     )
{
//	int start = K[offK];
/*
	for (int i = K[offK]; i < N; i++) {
	    std::cout << indicesN[i] << '\n';
	}
*/
	// fused scan reduction
	auto begin0 = thrust::make_zip_iterator(thrust::make_tuple(d_Numerator.begin(), d_Numerator2.begin()));
	auto begin1 = thrust::make_zip_iterator(thrust::make_tuple(d_AccDenom.begin(), d_NWeight.begin()));
	if (formatType == INDICATOR) {
		DeviceFuse::ScanReduce(d_temp_storage_gh, temp_storage_bytes_gh, begin0, begin1, d_BlockGH, d_GH,
				TuplePlus(), Double2Plus(), compGradHessInd, N);
	} else {
		DeviceFuse::ScanReduce(d_temp_storage_gh, temp_storage_bytes_gh, begin0, begin1, d_BlockGH, d_GH,
				TuplePlus(), Double2Plus(), compGradHessNInd, N);
	}
	hipDeviceSynchronize();
}


template <typename RealType>
void CudaKernel<RealType>::computeGradientAndHessian1(thrust::device_vector<RealType>& d_Numerator,
                                                     thrust::device_vector<RealType>& d_Numerator2,
                                                     thrust::device_vector<RealType>& d_Denominator,
                                                     thrust::device_vector<RealType>& d_AccNumer,
                                                     thrust::device_vector<RealType>& d_AccNumer2,
                                                     thrust::device_vector<RealType>& d_AccDenom,
                                                     thrust::device_vector<RealType>& d_NWeight,
                                                     double2* d_GH,
                                                     double2* d_BlockGH,
                                                     FormatType& formatType,
                                                     size_t& N
)
{
	// fused scan reduction
	auto begin2 = thrust::make_zip_iterator(thrust::make_tuple(d_Numerator.begin(),
				d_Numerator2.begin(),
				d_Denominator.begin()));
	if (formatType == INDICATOR) {

		DeviceFuse::ScanReduce(d_temp_storage_gh, temp_storage_bytes_gh,
                        begin2, thrust::raw_pointer_cast(&d_NWeight[0]),
                        d_BlockGH, d_GH,
                        TuplePlus3(), Double2Plus(), compGradHessInd1, N);
/*
		DeviceFuse::ScanReduce1(d_temp_storage_gh, temp_storage_bytes_gh,
				begin2, thrust::raw_pointer_cast(&d_NWeight[0]),
				d_BlockGH, d_GH, thrust::raw_pointer_cast(&d_AccDenom[0]),
				TuplePlus3(), Double2Plus(), compGradHessInd1, scanOutput, N);
*/
	} else {

		DeviceFuse::ScanReduce(d_temp_storage_gh, temp_storage_bytes_gh,
                        begin2, thrust::raw_pointer_cast(&d_NWeight[0]),
                        d_BlockGH, d_GH,
                        TuplePlus3(), Double2Plus(), compGradHessNInd1, N);
/*
		DeviceFuse::ScanReduce1(d_temp_storage_gh, temp_storage_bytes_gh,
				begin2, thrust::raw_pointer_cast(&d_NWeight[0]),
				d_BlockGH, d_GH, thrust::raw_pointer_cast(&d_AccDenom[0]),
				TuplePlus3(), Double2Plus(), compGradHessNInd1, scanOutput, N);
*/
	}

	hipDeviceSynchronize();
}


template <typename RealType, FormatTypeCuda formatType>
void dispatchPriorType(const thrust::device_vector<RealType>& d_X,
                        const thrust::device_vector<int>& d_K,
                        unsigned int offX,
                        unsigned int offK,
                        const unsigned int taskCount,
                        double2* d_GH,
                        thrust::device_vector<RealType>& d_XjY,
                        thrust::device_vector<RealType>& d_Bound,
                        thrust::device_vector<RealType>& d_KWeight,
                        thrust::device_vector<RealType>& d_Beta,
                        thrust::device_vector<RealType>& d_XBeta,
                        thrust::device_vector<RealType>& d_ExpXBeta,
                        thrust::device_vector<RealType>& d_Denominator,
                        thrust::device_vector<RealType>& d_Numerator,
                        thrust::device_vector<RealType>& d_Numerator2,
                        thrust::device_vector<RealType>& d_PriorParams,
                        const int priorTypes,
                        int index,
                        int gridSize, int blockSize)
{
	switch (priorTypes) {
		case 0 :
			kernelUpdateXBetaAndDelta<RealType, formatType, NOPRIOR><<<gridSize, blockSize>>>(offX, offK, taskCount, index,
                                                               thrust::raw_pointer_cast(&d_X[0]),
                                                               thrust::raw_pointer_cast(&d_K[0]),
                                                               d_GH,
                                                               thrust::raw_pointer_cast(&d_XjY[0]),
                                                               thrust::raw_pointer_cast(&d_Bound[0]),
                                                               thrust::raw_pointer_cast(&d_KWeight[0]),
                                                               thrust::raw_pointer_cast(&d_Beta[0]),
                                                               thrust::raw_pointer_cast(&d_XBeta[0]),
                                                               thrust::raw_pointer_cast(&d_ExpXBeta[0]),
                                                               thrust::raw_pointer_cast(&d_Denominator[0]),
                                                               thrust::raw_pointer_cast(&d_Numerator[0]),
                                                               thrust::raw_pointer_cast(&d_Numerator2[0]),
                                                               thrust::raw_pointer_cast(&d_PriorParams[0]));
			break;
		case 1 :
			kernelUpdateXBetaAndDelta<RealType, formatType, LAPLACE><<<gridSize, blockSize>>>(offX, offK, taskCount, index,
                                                               thrust::raw_pointer_cast(&d_X[0]),
                                                               thrust::raw_pointer_cast(&d_K[0]),
                                                               d_GH,
                                                               thrust::raw_pointer_cast(&d_XjY[0]),
                                                               thrust::raw_pointer_cast(&d_Bound[0]),
                                                               thrust::raw_pointer_cast(&d_KWeight[0]),
                                                               thrust::raw_pointer_cast(&d_Beta[0]),
                                                               thrust::raw_pointer_cast(&d_XBeta[0]),
                                                               thrust::raw_pointer_cast(&d_ExpXBeta[0]),
                                                               thrust::raw_pointer_cast(&d_Denominator[0]),
                                                               thrust::raw_pointer_cast(&d_Numerator[0]),
                                                               thrust::raw_pointer_cast(&d_Numerator2[0]),
                                                               thrust::raw_pointer_cast(&d_PriorParams[0]));
			break;
		case 2 :
			kernelUpdateXBetaAndDelta<RealType, formatType, NORMAL><<<gridSize, blockSize>>>(offX, offK, taskCount, index,
                                                               thrust::raw_pointer_cast(&d_X[0]),
                                                               thrust::raw_pointer_cast(&d_K[0]),
                                                               d_GH,
                                                               thrust::raw_pointer_cast(&d_XjY[0]),
                                                               thrust::raw_pointer_cast(&d_Bound[0]),
                                                               thrust::raw_pointer_cast(&d_KWeight[0]),
                                                               thrust::raw_pointer_cast(&d_Beta[0]),
                                                               thrust::raw_pointer_cast(&d_XBeta[0]),
                                                               thrust::raw_pointer_cast(&d_ExpXBeta[0]),
                                                               thrust::raw_pointer_cast(&d_Denominator[0]),
                                                               thrust::raw_pointer_cast(&d_Numerator[0]),
                                                               thrust::raw_pointer_cast(&d_Numerator2[0]),
                                                               thrust::raw_pointer_cast(&d_PriorParams[0]));
			break;
	}
}


template <typename RealType>
void CudaKernel<RealType>::updateXBetaAndDelta(const thrust::device_vector<RealType>& d_X,
                                       const thrust::device_vector<int>& d_K,
                                       unsigned int offX,
                                       unsigned int offK,
                                       const unsigned int taskCount,
                                       double2* d_GH,
                                       thrust::device_vector<RealType>& d_XjY,
                                       thrust::device_vector<RealType>& d_Bound,
                                       thrust::device_vector<RealType>& d_KWeight,
                                       thrust::device_vector<RealType>& d_Beta,
                                       thrust::device_vector<RealType>& d_XBeta,
                                       thrust::device_vector<RealType>& d_ExpXBeta,
                                       thrust::device_vector<RealType>& d_Denominator,
                                       thrust::device_vector<RealType>& d_Numerator,
                                       thrust::device_vector<RealType>& d_Numerator2,
                                       thrust::device_vector<RealType>& d_PriorParams,
                                       const int priorTypes,
                                       int index,
                                       FormatType& formatType,
                                       int gridSize, int blockSize)
{
	switch (formatType) {
		case DENSE :
			dispatchPriorType<RealType, DENSE>(d_X, d_K, offX, offK,
                                                          taskCount, d_GH, d_XjY, d_Bound, d_KWeight,
                                                          d_Beta, d_XBeta, d_ExpXBeta, d_Denominator,
                                                          d_Numerator, d_Numerator2,
                                                          d_PriorParams, priorTypes,
                                                          index, gridSize, blockSize);
			break;
		case SPARSE :
			dispatchPriorType<RealType, SPARSE>(d_X, d_K, offX, offK,
                                                          taskCount, d_GH, d_XjY, d_Bound, d_KWeight,
                                                          d_Beta, d_XBeta, d_ExpXBeta, d_Denominator,
                                                          d_Numerator, d_Numerator2,
                                                          d_PriorParams, priorTypes,
                                                          index, gridSize, blockSize);
			break;
		case INDICATOR :
			dispatchPriorType<RealType, INDICATOR>(d_X, d_K, offX, offK,
                                                          taskCount, d_GH, d_XjY, d_Bound, d_KWeight,
                                                          d_Beta, d_XBeta, d_ExpXBeta, d_Denominator,
                                                          d_Numerator, d_Numerator2,
                                                          d_PriorParams, priorTypes,
                                                          index, gridSize, blockSize);
			break;
		case INTERCEPT :
			dispatchPriorType<RealType, INTERCEPT>(d_X, d_K, offX, offK,
                                                          taskCount, d_GH, d_XjY, d_Bound, d_KWeight,
                                                          d_Beta, d_XBeta, d_ExpXBeta, d_Denominator,
                                                          d_Numerator, d_Numerator2,
                                                          d_PriorParams, priorTypes,
                                                          index, gridSize, blockSize);
			break;
	}

        hipDeviceSynchronize();
}


template <typename RealType>
void CudaKernel<RealType>::processDelta(double2* d_GH,
		thrust::device_vector<RealType>& d_XjY,
		thrust::device_vector<RealType>& d_Delta,
		thrust::device_vector<RealType>& d_Beta,
		thrust::device_vector<RealType>& d_Bound,
		thrust::device_vector<RealType>& d_PriorParams,
		const int priorType,
		int index,
		int gridSize, int blockSize)
{
	switch (priorType) {
		case 0 :
			kernelProcessDelta<RealType, NOPRIOR><<<gridSize, blockSize>>>(d_GH,
			        thrust::raw_pointer_cast(&d_XjY[0]),
			        thrust::raw_pointer_cast(&d_Delta[0]),
			        thrust::raw_pointer_cast(&d_Beta[0]),
			        thrust::raw_pointer_cast(&d_Bound[0]),
			        thrust::raw_pointer_cast(&d_PriorParams[0]),
			        index);
			break;
		case 1 :
			kernelProcessDelta<RealType, LAPLACE><<<gridSize, blockSize>>>(d_GH,
			        thrust::raw_pointer_cast(&d_XjY[0]),
			        thrust::raw_pointer_cast(&d_Delta[0]),
			        thrust::raw_pointer_cast(&d_Beta[0]),
			        thrust::raw_pointer_cast(&d_Bound[0]),
			        thrust::raw_pointer_cast(&d_PriorParams[0]),
			        index);
			break;
		case 2 :
			kernelProcessDelta<RealType, NORMAL><<<gridSize, blockSize>>>(d_GH,
			        thrust::raw_pointer_cast(&d_XjY[0]),
			        thrust::raw_pointer_cast(&d_Delta[0]),
			        thrust::raw_pointer_cast(&d_Beta[0]),
			        thrust::raw_pointer_cast(&d_Bound[0]),
			        thrust::raw_pointer_cast(&d_PriorParams[0]),
			        index);
			break;
	}

    hipDeviceSynchronize();
}

template <typename RealType>
void CudaKernel<RealType>::updateXBeta(const thrust::device_vector<RealType>& d_X,
		const thrust::device_vector<int>& d_K,
		unsigned int offX,
		unsigned int offK,
		const unsigned int taskCount,
		thrust::device_vector<RealType>& d_Delta,
		thrust::device_vector<RealType>& d_KWeight,
		thrust::device_vector<RealType>& d_XBeta,
		thrust::device_vector<RealType>& d_ExpXBeta,
		thrust::device_vector<RealType>& d_Denominator,
		thrust::device_vector<RealType>& d_Numerator,
		thrust::device_vector<RealType>& d_Numerator2,
		int index,
		FormatType& formatType,
		int gridSize, int blockSize)
{
	switch (formatType) {
		case DENSE :
			kernelUpdateXBeta<RealType, DENSE><<<gridSize, blockSize>>>(offX, offK, taskCount, d_Delta[index],
			        thrust::raw_pointer_cast(&d_X[0]),
			        thrust::raw_pointer_cast(&d_K[0]),
			        thrust::raw_pointer_cast(&d_KWeight[0]),
			        thrust::raw_pointer_cast(&d_XBeta[0]),
			        thrust::raw_pointer_cast(&d_ExpXBeta[0]),
			        thrust::raw_pointer_cast(&d_Denominator[0]),
			        thrust::raw_pointer_cast(&d_Numerator[0]),
			        thrust::raw_pointer_cast(&d_Numerator2[0]));
			break;
		case SPARSE :
			kernelUpdateXBeta<RealType, SPARSE><<<gridSize, blockSize>>>(offX, offK, taskCount, d_Delta[index],
			        thrust::raw_pointer_cast(&d_X[0]),
			        thrust::raw_pointer_cast(&d_K[0]),
			        thrust::raw_pointer_cast(&d_KWeight[0]),
			        thrust::raw_pointer_cast(&d_XBeta[0]),
			        thrust::raw_pointer_cast(&d_ExpXBeta[0]),
			        thrust::raw_pointer_cast(&d_Denominator[0]),
			        thrust::raw_pointer_cast(&d_Numerator[0]),
			        thrust::raw_pointer_cast(&d_Numerator2[0]));
			break;
		case INDICATOR :
			kernelUpdateXBeta<RealType, INDICATOR><<<gridSize, blockSize>>>(offX, offK, taskCount, d_Delta[index],
			        thrust::raw_pointer_cast(&d_X[0]),
			        thrust::raw_pointer_cast(&d_K[0]),
			        thrust::raw_pointer_cast(&d_KWeight[0]),
			        thrust::raw_pointer_cast(&d_XBeta[0]),
			        thrust::raw_pointer_cast(&d_ExpXBeta[0]),
			        thrust::raw_pointer_cast(&d_Denominator[0]),
			        thrust::raw_pointer_cast(&d_Numerator[0]),
			        thrust::raw_pointer_cast(&d_Numerator2[0]));
			break;
		case INTERCEPT :
			kernelUpdateXBeta<RealType, INTERCEPT><<<gridSize, blockSize>>>(offX, offK, taskCount, d_Delta[index],
			        thrust::raw_pointer_cast(&d_X[0]),
			        thrust::raw_pointer_cast(&d_K[0]),
			        thrust::raw_pointer_cast(&d_KWeight[0]),
			        thrust::raw_pointer_cast(&d_XBeta[0]),
			        thrust::raw_pointer_cast(&d_ExpXBeta[0]),
			        thrust::raw_pointer_cast(&d_Denominator[0]),
			        thrust::raw_pointer_cast(&d_Numerator[0]),
			        thrust::raw_pointer_cast(&d_Numerator2[0]));
			break;
	}

	hipDeviceSynchronize();
}


template <typename RealType>
void CudaKernel<RealType>::empty4(thrust::device_vector<RealType>& d_AccNumer,
				  thrust::device_vector<RealType>& d_AccNumer2,
				  thrust::device_vector<RealType>& d_Buffer1,
				  thrust::device_vector<RealType>& d_Buffer2)
{
	d_Buffer1 = d_AccNumer;
	d_Buffer2 = d_AccNumer2;
}

template <typename RealType>
void CudaKernel<RealType>::empty2(thrust::device_vector<RealType>& d_AccDenom,
                                  thrust::device_vector<RealType>& d_Buffer3)
{
	d_Buffer3 = d_AccDenom;
}


template <typename RealType>
void CudaKernel<RealType>::CubScan(RealType* d_in, RealType* d_out, int num_items)
{
	// Launch kernel
	DeviceScan::InclusiveSum(d_temp_storage0, temp_storage_bytes0, d_in, d_out, num_items);
	hipDeviceSynchronize(); // MAS Wait until kernel completes; may be important for timing
}


/* currently not using

template <typename RealType>
void CudaKernel<RealType>::computeAccumulatedNumerator(thrust::device_vector<RealType>& d_Numerator,
                                                       thrust::device_vector<RealType>& d_Numerator2,
                                                       thrust::device_vector<RealType>& d_AccNumer,
                                                       thrust::device_vector<RealType>& d_AccNumer2,
                                                       size_t& N)
{
        auto results = thrust::make_zip_iterator(thrust::make_tuple(d_AccNumer.begin(), d_AccNumer2.begin()));
        auto begin = thrust::make_zip_iterator(thrust::make_tuple(d_Numerator.begin(), d_Numerator2.begin()));

        // Launch kernel
        DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, begin, results, TuplePlus(), N);
        hipDeviceSynchronize(); // MAS Wait until kernel completes; may be important for timing
}

template <typename RealType>
void CudaKernel<RealType>::computeAccumulatedNumerAndDenom(thrust::device_vector<RealType>& d_Denominator,
                                                           thrust::device_vector<RealType>& d_Numerator,
                                                           thrust::device_vector<RealType>& d_Numerator2,
                                                           thrust::device_vector<RealType>& d_AccDenom,
                                                           thrust::device_vector<RealType>& d_AccNumer,
                                                           thrust::device_vector<RealType>& d_AccNumer2,
                                                           size_t& N)
{
        auto results_acc = thrust::make_zip_iterator(thrust::make_tuple(d_AccDenom.begin(), d_AccNumer.begin(), d_AccNumer2.begin()));
        auto begin_acc = thrust::make_zip_iterator(thrust::make_tuple(d_Denominator.begin(), d_Numerator.begin(), d_Numerator2.begin()));

        // Launch kernel
        DeviceScan::InclusiveScan(d_temp_storage_acc, temp_storage_bytes_acc, begin_acc, results_acc, TuplePlus3(), N);
        hipDeviceSynchronize(); // MAS Wait until kernel completes; may be important for timing
}

*/

template class CudaKernel<float>;
template class CudaKernel<double>;

